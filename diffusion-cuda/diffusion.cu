#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define OS WIN32

#define WIN32 1
#define LINUX 2

#if (OS == WIN32)
#include <time.h>
#elif (OS == LINUX)
#include <sys/time.h>
#endif

#define DISPLAY_STEP (1)
#define SELECT_WAY (1) // 1
#define EN_VERIFY (0)  // 0/1
#define NX 8192
#define NY 8192
#define BS (32) //(32,32,1)/(16,16,1)/(8,8,1)/(4,4,1)

float* data = NULL;
float* gpu_data = NULL;

// For precise time measurement
#if (OS == WIN32)
clock_t st, st2, et2, et;
#elif (OS == LINUX)
struct timeval st, st2, et2, et;
#endif

#if (OS == WIN32)
/* in microseconds (us) */
double get_elapsed_time(clock_t *begin, clock_t *end)
{
  return ((double)(*end - *begin)) / CLOCKS_PER_SEC * 1000000;
}
#elif (OS == LINUX)
/* in microseconds (us) */
double get_elapsed_time(struct timeval *begin, struct timeval *end)
{
  return (end->tv_sec - begin->tv_sec) * 1000000 + (end->tv_usec - begin->tv_usec);
}
#endif

void init(int nx, int ny, float *data)
{
  int x, y;
  int cx = nx / 2, cy = 0; /* center of ink */
  int rad = (nx + ny) / 8; /* radius of ink */
  int block = nx * ny;

  for (y = 0; y < ny; y++)
  {
    for (x = 0; x < nx; x++)
    {
      float v = 0.0;
      if (((x - cx) * (x - cx) + (y - cy) * (y - cy)) < rad * rad)
      {
        v = 1.0; // They represent the ink's region.
      }
      data[0 + y * nx + x] = v;
      data[1 * block + y * nx + x] = v;
    }
  }
  return;
}

__global__ void calculation_onestep_kernel(int t, int nx, int ny, int block, float* gpu_data)
{
  // An array for “even” steps, An array for “odd” steps. (i.e., “current” array and “next” array)
  int from = t % 2;
  int to = (t + 1) % 2;
  int i, j;

  j = blockIdx.y * blockDim.y + threadIdx.y;
  i = blockIdx.x * blockDim.x + threadIdx.x;

  // Ignore boundary data
  if ((i == 0) || (j == 0) || (i >= nx - 1) || (j >= ny - 1))
  {
    return;
  }

#define x i
#define y j
  gpu_data[to * block + y * nx + x] = 0.2 * (gpu_data[from * block + y * nx + x] + gpu_data[from * block + y * nx + x - 1] \
  + gpu_data[from * block + y * nx + x + 1] + gpu_data[from * block + (y - 1) * nx + x] + gpu_data[from * block + (y + 1) * nx + x]);
#undef x
#undef y
}

/* Calculate for one time step */
/* Input: data[t%2], Output: data[(t+1)%2] */
void calc(int nt, int nx, int ny, float *data)
{
  int t;
  int block = nx * ny;
  hipError_t rc;
  int data_size = nx * ny * 2;
  int result = nt % 2;

  int temp = 0;
  temp = (int) ceil(sqrt(block / (BS * BS)));
  if (temp > USHRT_MAX)
  {
    printf("Error: block is too big!\n");
    exit(1);
  }
  else if (!temp) // One grid is enough.
    temp = 1;
  printf("temp is[%d].\n", temp);
  dim3 gpu_grid = dim3(temp, temp, 1);
  dim3 gpu_block = dim3(BS, BS, 1);

  /* allocate device memory */
  rc = hipMalloc((void **)&gpu_data, sizeof(float) * data_size); // Double buffers.
  if (rc != hipSuccess)
  {
    printf("Error: hipMalloc failed.\n");
    exit(1);
  }

#if (OS == WIN32)
  st = clock();
#elif (OS == LINUX)
  gettimeofday(&st, NULL);
#endif
  /* copy input data from host to device */
  hipMemcpy(gpu_data, data, sizeof(float) * data_size, hipMemcpyHostToDevice);
  hipDeviceSynchronize(); /* for precise time measurement */

#if (OS == WIN32)
  st2 = clock();
#elif (OS == LINUX)
  gettimeofday(&st2, NULL);
#endif

  /* computation */
  for (t = 0; t < nt; t++)
  {
#if DISPLAY_STEP
    printf("step %d\n", t);
    //fflush(0);
#endif

    calculation_onestep_kernel<<<gpu_grid, gpu_block>>>(t, nx, ny, block, gpu_data);
  }
  hipDeviceSynchronize(); /* for precise time measurement */

#if (OS == WIN32)
  et2 = clock();
#elif (OS == LINUX)
  gettimeofday(&et2, NULL);
#endif

  /* copy output data (only result data) from device to host */
  hipMemcpy(&(data[result * block]), &(gpu_data[result * block]), sizeof(float) * data_size / 2, hipMemcpyDeviceToHost);
  hipDeviceSynchronize(); /* for precise time measurement */

#if (OS == WIN32)
  et = clock();
#elif (OS == LINUX)
  gettimeofday(&et, NULL);
#endif

  hipFree(gpu_data);
  return;
}

int main(int argc, char *argv[])
{
  int nt = 20; /* number of time steps */
  int nx = NX, ny = NY;

  printf("Function is %s, BS=[%d].\n", argv[0], BS);
  if (argc >= 2)
  { /* if an argument is specified */
    nt = atoi(argv[1]);
    printf("nt is %d, ", nt);
    if (argc >= 4)
    {
      nx = atoi(argv[2]); // < 65536
      ny = atoi(argv[3]);
      printf("nx is %d, ny is %d. (Only accept equal numbers: nx=ny)\n", nx, ny);
      if ((nx > USHRT_MAX) || (ny > USHRT_MAX))
      {
        printf("Error: nx or ny is too big!\n");
        exit(1);
      }
      if (nx != ny)
      {
        printf("Error: nx is not equal to ny!\n");
        exit(1);
      }
    }
  }

  data = (float *)malloc(sizeof(float) * nx * ny * 2); // Double buffers.

  if (data == NULL)
  {
    printf("Error: Memory allocation failed.\n");
    exit(1);
  }

  init(nx, ny, data);

#if (OS == WIN32)
  clock_t start, stop;
  start = clock();

  calc(nt, nx, ny, data);

  stop = clock();

  {
    double us;
    double gflops;
    int op_per_point = 5; // 4 add & 1 multiply per point and some integer operation

    us = get_elapsed_time(&start, &stop);
    printf("Elapsed time: %.3lf sec\n", us / 1000000.0);
    gflops = ((double)nx * ny * nt * op_per_point) / us / 1000.0;
    printf("Speed: %.3lf GFlops\n", gflops);

    double us2;
    us = get_elapsed_time(&st, &et);
    us2 = get_elapsed_time(&st2, &et2);
    gflops = ((double)nx * ny * nt * op_per_point) / us / 1000.0;
    printf("Calculation took %.3lf sec --> %.3lf GFlops  (with data transfer)\n",
           us / 1000000.0, gflops);
    gflops = ((double)nx * ny * nt * op_per_point) / us2 / 1000.0;
    printf("                 %.3lf sec --> %.3lf GFlops  (without data transfer)\n",
           us2 / 1000000.0, gflops);
  }

#elif (OS == LINUX)
  struct timeval t1, t2;
  gettimeofday(&t1, NULL);

  calc(nt, nx, ny, data);

  gettimeofday(&t2, NULL);

  {
    double us;
    double gflops;
    int op_per_point = 5; // 4 add & 1 multiply per point

    us = get_elapsed_time(&t1, &t2);
    printf("Elapsed time: %.3lf sec\n", us / 1000000.0);
    gflops = ((double)nx * ny * nt * op_per_point) / us / 1000.0;
    printf("Speed: %.3lf GFlops\n", gflops);

    double us2;
    us = get_elapsed_time(&st, &et);
    us2 = get_elapsed_time(&st2, &et2);
    gflops = ((double)nx * ny * nt * op_per_point) / us / 1000.0;
    printf("Calculation took %.3lf sec --> %.3lf GFlops  (with data transfer)\n",
           us / 1000000.0, gflops);
    gflops = ((double)nx * ny * nt * op_per_point) / us2 / 1000.0;
    printf("                 %.3lf sec --> %.3lf GFlops  (without data transfer)\n",
           us2 / 1000000.0, gflops);
  }
#endif

#if (EN_VERIFY == 1)
  int result = nt % 2;
  int block = nx * ny;

  /* Store the result generated by parallelism  . */
#if (SELECT_WAY == 1)
  FILE *fd1 = fopen("Data_w1", "wb");
  size_t num1 = fwrite(data + result * block, sizeof(float), block, fd1);
  printf("VERIFY: fd1 writes [%ld] elements.\n", (long)num1);
  fclose(fd1);

#endif

  /* Read the result that is created by “diffusion” sample program */
  FILE *fd2 = fopen("Data", "rb");
  //FILE *fd2 = fopen("Data_w1", "rb");
  if (fd2 == NULL)
  {
    printf("VERIFY Error: Open file Data.txt failed.\n");
    exit(1);
  }
  int test = (nt + 1) % 2;
  size_t num2 = fread(data + test * block, sizeof(float), block, fd2);
  printf("VERIFY: fd2 reads [%ld] elements.\n", (long)num2);

  /* Below we compare the results. */
  unsigned char error_flag = 0;
  int y;
  for (y = 1; y < ny - 1; y++)
  {
    int x;
    for (x = 1; x < nx - 1; x++)
    {
      if (data[test * block + y * nx + x] != data[result * block + y * nx + x])
      {
        error_flag = 1;
        printf("VERIFY: We got error flag at y = [%d] and x = [%d]!\n", x, y);
        break;
      }
    }
  }

  if (!error_flag)
    printf("VERIFY: Succeed! We have no error flag!\n");
#endif

  free(data);
  return 0;
}
